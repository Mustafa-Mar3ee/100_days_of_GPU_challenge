#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
    {
        c[index] = a[index] + b[index];
    }
}

int main()
{
    int N = 1000;
    int size = N * sizeof(int);
    int *h_a, *h_b, *h_c;
    hipHostMalloc(&h_a, size, hipHostMallocDefault);
    hipHostMalloc(&h_b, size, hipHostMallocDefault);
    hipHostMalloc(&h_c, size, hipHostMallocDefault);

    for (int i = 0; i < N; i++)
    {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate memory on GPU
    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy Data from HOST (CPU) to Device (GPU)

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blockPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vectorAdd<<<blockPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    printf("Vector Addition Rsults (First 10 Elements):\n");
    for (int i = 0; i < 10; i++)
    {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipFree(h_b);
    hipFree(h_c);

    return 0;
}