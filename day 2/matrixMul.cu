#include <iostream>
#include <hip/hip_runtime.h>

#define N 2 // Matrix size (N x N)

// CUDA kernel for matrix multiplication
__global__ void matrixMul(int *A, int *B, int *C, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n)
    {
        int sum = 0;
        for (int k = 0; k < n; k++)
        {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main()
{

    int size = N * N * sizeof(int);

    // Host matrices
    int h_A[N * N] = {1, 2, 3, 4};
    int h_B[N * N] = {5, 6, 7, 8};
    int h_C[N * N];

    // Device matrices
    int *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define grid and block size
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    // Launch kernel
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result Matrix C:\n";
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free memory
    // Free Device Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free Host Memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
